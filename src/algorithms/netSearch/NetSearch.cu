#include "hip/hip_runtime.h"
/**
 * File        : NetSearch.cu
 * Description : Implementation of the novel algorithm, NetSearch, using CUDA.
 */

#include <cstdlib>
#include <iostream>
#include <fstream>
#include <sstream>
#include <unordered_map>
#include <vector>
#include <cmath>
#include <chrono>

#include <boost/lexical_cast.hpp>

#include <hip/hip_runtime.h>

#include "algorithms/tools/PathTile.h"
#include "common/Results.h"

 using namespace pathFind;

const std::string WORLD_DIR = "../worlds";
const std::string WORLD_EXT = ".world";
const std::string PATH_EXT = ".path";

const std::string ALG_NAME = "netSearch";

const size_t BLOCK_WIDTH = 256;
const size_t BLOCK_HEIGHT = 256;

struct ThreadTile
{
    struct Branch
    {
        bool finished = false;
        uint64_t bestCost = std::numeric_limits<uint64_t>::max();
    };

    uint64_t cost = 0;
    Branch bestBranch;
    Branch branches[4];
};

__global__ void
netSearchIteration (ThreadTile* tiles, size_t pitch, size_t width, size_t height)
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < width && idy < height)
    {
        uint64_t pos = idx + idy * pitch;
        if (tiles[pos].cost != 0 && !tiles[pos].bestBranch.finished)
        {
            uint64_t neighborsX[4] = {
                idx,
                idx + 1,
                idx,
                idx - 1
            };

            uint64_t neighborsY[4] = {
                idy - 1,
                idy,
                idy + 1,
                idy
            };

            for (uint i = 0; i < 4; ++i)
            {

                if (neighborsX[i] < width && neighborsY[i] < height)
                {
                    uint64_t neighbor = neighborsX[i] + neighborsY[i] * pitch;
                    tiles[pos].branches[i].bestCost = tiles[neighbor].bestBranch.bestCost + tiles[pos].cost;
                    tiles[pos].branches[i].finished = tiles[neighbor].bestBranch.finished;
                }
                tiles[pos].branches[i].finished = true;
            }
        }
    }
}

__global__ void
threadUpdate (bool* finished, ThreadTile* tiles, size_t pitch, size_t width, size_t height, uint64_t startX, uint64_t startY)
{
    uint idx = blockIdx.x * blockDim.x + threadIdx.x;
    uint idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < width && idy < height)
    {
        uint64_t pos = idx + idy * pitch;
        if (tiles[pos].cost != 0 && !tiles[pos].bestBranch.finished)
        {
            uint64_t min = std::numeric_limits<uint64_t>::max();
            uint64_t minIndex = 4;
            for (uint i = 0; i < 4; ++i)
            {
                if (tiles[pos].branches[i].bestCost < min)
                {
                    min = tiles[pos].branches[i].bestCost;
                    minIndex = i;
                }
            }
            tiles[pos].bestBranch = tiles[pos].branches[minIndex];

            // If start tile knows best path then we can notify the CPU
            if (idx == startX && idy == startY && tiles[pos].bestBranch.finished)
            {
                *finished = true;
            }

        }
    }
}

int main (int args, char* argv[])
{
    // Program should be started with 5 command line parameters (or 1)
    // that specifies the name of the world file to read from and then optionallys
    // the start x, start y, end x, and end y
    if (args != 6  && args != 2)
    {
        std::cout << "Incorrect inputs. Usage: <filename> (start x) (start y) (end x) (end y)" << std::endl;
        return EXIT_FAILURE;
    }

    // Parse the world file
    std::stringstream filename;
    filename << WORLD_DIR << "/" << argv[1] << WORLD_EXT;

    std::ifstream worldFile (filename.str (),
            std::ifstream::in | std::ifstream::binary);

    if (!worldFile)
    {
        std::cout << "World file doesn't exist." << std::endl;
        return EXIT_FAILURE;
    }
    pathFind::World world;

    worldFile >> world;
    worldFile.close ();

    uint startX, startY, endX, endY;

    if (args == 6)
    {
        // Parse the start and end points
        try
        {
            startX = boost::lexical_cast<uint> (argv[2]);
            startY = boost::lexical_cast<uint> (argv[3]);
            endX = boost::lexical_cast<uint> (argv[4]);
            endY = boost::lexical_cast<uint> (argv[5]);
        } catch (boost::bad_lexical_cast &e)
        {
            std::cout << "Start and end points failed to convert to numeric types" << std::endl;
            return EXIT_FAILURE;
        }
    }
    else
    {
        std::stringstream pathFilename;
        pathFilename << WORLD_DIR << "/" << argv[1] << PATH_EXT;
        std::ifstream pathIn (pathFilename.str ());
        if (!pathIn)
        {
            std::string pathCommand = "./pathGen " + std::string (argv[1]);
            system (pathCommand.c_str());
            pathIn.close ();
            pathIn.open (pathFilename.str ());
            if (!pathIn)
            {
                std::cout << "Could not construct path." << std::endl;
                return EXIT_FAILURE;
            }
        }
        pathIn >> startX >> startY >> endX >> endY;

    }

    // Intialize Cuda with flag that allows mapped memory access on device
    hipSetDeviceFlags (hipDeviceMapHost);

    std::vector<ThreadTile> h_threadTiles (world.getWidth() * world.getHeight());
    for (size_t x = 0; x < world.getWidth(); ++x)
    {
        for (size_t y = 0; y < world.getHeight(); ++y)
        {
            uint64_t bestCost = world (x, y).cost;
            h_threadTiles[x + y * world.getWidth ()].cost = bestCost;
            if (bestCost == 0)
            {
                bestCost = std::numeric_limits<uint64_t>::max();
            }
            else
            {
                bestCost += (x < endX ? endX - x : x - endX) +
                            (y < endY ? endY - y : y - endY);
            }
            h_threadTiles[x + y * world.getWidth ()].bestBranch.bestCost = bestCost;
        }
    }

    auto t1 = std::chrono::high_resolution_clock::now();

    h_threadTiles[endX + endY * world.getWidth ()].bestBranch.finished = true;
    h_threadTiles[endX + endY * world.getWidth ()].bestBranch.bestCost = 0;

    ThreadTile* d_threadTiles;
    size_t pitch;
    hipMallocPitch (reinterpret_cast<void **> (&d_threadTiles), &pitch, world.getWidth () * sizeof(ThreadTile), world.getHeight ());
    hipMemcpy2D (d_threadTiles, pitch, h_threadTiles.data (), world.getWidth () * sizeof (ThreadTile),
            world.getWidth () * sizeof (ThreadTile), world.getHeight (), hipMemcpyHostToDevice);

    // Set up mapped page-locked memory so that the gpu can quickly tell the cpu when a path is found
    bool* h_finished;
    bool* d_finished;
    hipHostAlloc (reinterpret_cast<void**> (&h_finished), sizeof (bool), hipHostMallocMapped);
    //hipMalloc (reinterpret_cast<void**> (&d_finished), sizeof (bool))
    *h_finished = false;
    hipHostGetDevicePointer (reinterpret_cast<void**> (&d_finished), h_finished, 0);


    dim3 blockSize (BLOCK_WIDTH, BLOCK_HEIGHT);

    // Figure out how many blocks we will need based on how many tiles there are in the world
    size_t gridWidth = (world.getWidth () % BLOCK_WIDTH == 0) ?
                        world.getWidth () / BLOCK_WIDTH :
                        world.getWidth () / BLOCK_WIDTH + 1;
    size_t gridHeight = (world.getHeight () % BLOCK_HEIGHT == 0) ?
                        world.getHeight () / BLOCK_HEIGHT :
                        world.getHeight () / BLOCK_HEIGHT + 1;
    dim3 gridSize (gridWidth, gridHeight);

    uint64_t iter = 0;
    //while (!(*h_finished) && iter < 100000)
    //{
        netSearchIteration <<< gridSize, blockSize >>> (d_threadTiles, pitch, world.getWidth (), world.getHeight ());
        //threadUpdate <<< gridSize, blockSize >>> (d_finished, d_threadTiles, pitch, world.getWidth (), world.getHeight (), startX, startY);
        //hipDeviceSynchronize ();
        //std::cout << "iter: " << iter << std::endl;
        ++iter;
    //}

    hipMemcpy2D (h_threadTiles.data (), world.getWidth () * sizeof (ThreadTile), d_threadTiles, pitch,
            world.getWidth () * sizeof(ThreadTile), world.getHeight (), hipMemcpyDeviceToHost);

    auto t2 = std::chrono::high_resolution_clock::now();

    hipHostFree (h_finished);
    hipFree (d_threadTiles);

    std::vector<Point> finalPath;
    Point tile {startX, startY};
    uint64_t totalCost = h_threadTiles[tile.x + tile.y * world.getWidth ()].bestBranch.bestCost -
                        h_threadTiles[tile.x + tile.y * world.getWidth ()].cost;

    std::cout << "end bestCost: " << h_threadTiles[endX + endY * world.getWidth ()].bestBranch.bestCost
            << " finished: " << h_threadTiles[endX + endY * world.getWidth ()].bestBranch.finished << std::endl;

    std::cout << "endx - 1 bestCost: " << h_threadTiles[endX - 1 + endY * world.getWidth ()].bestBranch.bestCost
            << " finished: " << h_threadTiles[endX - 1 + endY * world.getWidth ()].bestBranch.finished << std::endl;

    std::cout << "endx - 1 branch bestCost: " << h_threadTiles[endX - 1 + endY * world.getWidth ()].branches[1].bestCost
            << " finished: " << h_threadTiles[endX - 1 + endY * world.getWidth ()].branches[1].finished << std::endl;

    std::cout << "endy - 1 bestCost: " << h_threadTiles[endX + (endY - 1) * world.getWidth ()].bestBranch.bestCost
            << " finished: " << h_threadTiles[endX + (endY - 1) * world.getWidth ()].bestBranch.finished << std::endl;
    while (tile.x != endX || tile.y != endY)
    {
        std::cout << "x: " << tile.x << " y: " << tile.y << std::endl;
        finalPath.emplace_back (tile);
        uint64_t id = tile.x + tile.y * world.getWidth ();
        uint64_t bestCost = h_threadTiles[id].bestBranch.bestCost;
        std::cout << "bestCost: " << bestCost << " finished: " << h_threadTiles[id].bestBranch.finished << "\n\n";
        uint64_t bestTile = 4;
        for (uint i = 0; i < 4; ++i)
        {
            if (h_threadTiles[id].branches[i].finished && h_threadTiles[id].branches[i].bestCost == bestCost)
            {
                bestTile = i;
                break;
            }
        }
        switch (bestTile)
        {
        case 0:
            tile.y--;
            break;
        case 1:
            tile.x++;
            break;
        case 2:
            tile.y++;
            break;
        case 3:
            tile.x--;
            break;
        }
    }

    std::vector<Point> reversedFinalPath (finalPath.rbegin (), finalPath.rend());
/*
    // Parse results into a stack
    uint totalCost = tile.getBestCost() - tile.getTile().cost;
    std::vector<Point> finalPath;
    while (tile.xy ().x != startX || tile.xy ().y != startY)
    {
        finalPath.emplace_back(tile.xy ());
        tile = expandedTiles[(tile.bestTile ().y * world.getWidth()) + tile.bestTile ().x];
    }
    finalPath.emplace_back(tile.xy ()); */

    writeResults (reversedFinalPath, argv[1], ALG_NAME,
            std::chrono::duration_cast<std::chrono::milliseconds>(t2-t1).count(), totalCost);

    return EXIT_SUCCESS;
}
